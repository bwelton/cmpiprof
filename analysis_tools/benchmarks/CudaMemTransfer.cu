
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %sn", 
            cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

int main(void)
{
  float tmp;
  size_t dataSizes[] = {100000 / 4,1000000 / 4,5000000 / 4,10000000 / 4,20000000 / 4,40000000 / 4,80000000 / 4,100000000/4};
  int count = 8;
  printf("Size of float: %u\n", sizeof(float));
  for (int i = 0; i < count; i++) {
  	float mallocTime, copyToDevice, copyFromDevice;


  	float * x = (float *) malloc(dataSizes[i]*sizeof(float));
  	memset(x, 1, dataSizes[i]*sizeof(float));
  	float * d_x;
  	{
	  	hipEvent_t start;
	  	hipEvent_t end;
	  	hipEventCreate(&start);
		hipEventCreate(&end);

		hipEventRecord(start);
		checkCuda(hipMalloc(&d_x, dataSizes[i]*sizeof(float)));
		hipEventRecord(end);
		hipEventSynchronize(end);
		hipEventElapsedTime(&mallocTime, start,end);
		hipEventDestroy(start);
		hipEventDestroy(end);
	}
	{
	  	hipEvent_t start;
	  	hipEvent_t end;
	  	hipEventCreate(&start);
		hipEventCreate(&end);

		hipEventRecord(start);
		checkCuda(hipMemcpy(d_x, x, dataSizes[i]*sizeof(float), hipMemcpyHostToDevice));
		hipEventRecord(end);
		hipEventSynchronize(end);
		hipEventElapsedTime(&copyToDevice, start,end);
		hipEventDestroy(start);
		hipEventDestroy(end);		
	} 
	{
	  	hipEvent_t start;
	  	hipEvent_t end;
	  	hipEventCreate(&start);
		hipEventCreate(&end);

		hipEventRecord(start);
		checkCuda(hipMemcpy(x,d_x, dataSizes[i]*sizeof(float), hipMemcpyDeviceToHost));
		hipEventRecord(end);
		hipEventSynchronize(end);
		hipEventElapsedTime(&copyFromDevice, start,end);
		hipEventDestroy(start);
		hipEventDestroy(end);	
	}
	hipFree(d_x);
	free(x);
	printf("%llu,%f,%f,%f\n",dataSizes[i] * 4,mallocTime/1000,copyToDevice/1000,copyFromDevice/1000);
  }
}